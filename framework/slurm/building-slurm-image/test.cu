#include <iostream>
#include <hip/hip_runtime.h>

__global__ void fibonacciGPU(int n, unsigned long long* result)
{
    int tid = threadIdx.x;

    if (tid == 0)
    {
        result[0] = 0; // First Fibonacci number
        result[1] = 1; // Second Fibonacci number

        for (int i = 2; i <= n; i++)
        {
            result[i] = result[i - 1] + result[i - 2];
        }
    }
}

int main()
{
    int n = 10; // Number of Fibonacci numbers to compute

    // Allocate memory on the GPU
    unsigned long long* d_result;
    hipMalloc((void**)&d_result, (n + 1) * sizeof(unsigned long long));

    // Launch the kernel on the GPU
    fibonacciGPU<<<1, 1>>>(n, d_result);

    // Copy the result back to the host
    unsigned long long* h_result = new unsigned long long[n + 1];
    hipMemcpy(h_result, d_result, (n + 1) * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Print the Fibonacci numbers
    for (int i = 0; i <= n; i++)
    {
        std::cout << h_result[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    delete[] h_result;
    hipFree(d_result);

    return 0;
}
